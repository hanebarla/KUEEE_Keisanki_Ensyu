
#include <hip/hip_runtime.h>
#include <iostream>
// Kernel detenition
__global__ void MatAdd(int N, float *A, float *B, float *C){
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[N*i + j] = A[N*i + j] + B[N*i + j];
}

int main(){
    float *A, *B, *C;
    int N = 100;
    hipMalloc((void**)&A, N*N*sizeof(float));
    hipMalloc((void**)&B, N*N*sizeof(float));
    hipMalloc((void**)&C, N*N*sizeof(float));
    auto *a = malloc(N*N*sizeof(float));
    auto *b = malloc(N*N*sizeof(float));
    auto *c = malloc(N*N*sizeof(float));
    hipMemcpy(A, a, N*N*sizeof(*A), hipMemcpyHostToDevice);
    hipMemcpy(B, b, N*N*sizeof(*B), hipMemcpyHostToDevice);

    // Kernel invocation with one block of N*N*1thread
    int numBlocks = 1;
    dim3 threadsPerBlock(N, N, 1);

    MatAdd <<< numBlocks, threadsPerBlock >>>(N, (float*)A, (float*)B, (float*)C);

    hipMemcpy(c, C, N*N*sizeof(*C), hipMemcpyDeviceToHost);

    hipFree(A);
    hipFree(B);
    hipFree(C);
    std::cout << "Done" << std::endl;
}